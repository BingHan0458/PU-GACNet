#include "hip/hip_runtime.h"  

#include "stdio.h"

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void isinblock_kernel(float *xy,  float x_low, float y_low, float size, bool *is_block, int N) {
 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < N){ 
        float x = xy[idx*2];
        float y = xy[idx*2+1];
        float x_up = x_low+size;
        float y_up = y_low+size;

        if (x>x_low && x<x_up && y>y_low && y<y_up)
            is_block[idx] = true;

        idx += blockDim.x*gridDim.x;
    }

}

extern "C" void is_in_block(float *xy_host, bool *is_block_host, float &x_low, float &y_low, float &size, int &N){
    float *xy;
    bool *is_block;
    hipError_t error;

    hipMalloc((void**)&xy, sizeof(float)* N*2);  
    hipMalloc((void**)&is_block, sizeof(bool)* N); 

    hipMemcpy(xy, xy_host, sizeof(float)* N*2, hipMemcpyHostToDevice);
    hipMemcpy(is_block, is_block_host, sizeof(bool)* N, hipMemcpyHostToDevice);

    isinblock_kernel<<<32786, 256>>>(xy, x_low, y_low, size, is_block, N);

    error = hipDeviceSynchronize();
    if(error != hipSuccess){
        printf("code: %d, reason: %s\n",error,hipGetErrorString(error));
    }

    hipMemcpy(is_block_host, is_block, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipFree(xy);
    hipFree(is_block);
}

